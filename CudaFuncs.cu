
#include <hip/hip_runtime.h>
/*
#include <cuda_runtime.h>
#include <string.h>
#include <stdio.h>

Mutant *setGPUforMutantCreation(Sequence *seq1, Sequence *seq2, int gpuSize, double *weights, int typeMutant)
{
    char *d_seq1, *d_seq2;
    double *d_weights;
    Mutant *gpuMutant;
    Mutant *tempMutant;

    int numOfBlocks = gpuSize;                      // each block 1 offset
    int NumOfThreads = (gpuSize * seq2->size) * 32; // change letters per index

    // Set the allocations for the device
    cudaMalloc((void **)&d_seq1, (seq1->size) * sizeof(char));
    cudaMalloc((void **)&d_seq2, (seq2->size) * sizeof(char));
    cudaMalloc((void **)&d_weights, 4 * sizeof(double));
    cudaMalloc((void **)&gpuMutant, (seq2->size) * sizeof(Mutant));
    cudaMalloc((void **)&tempMutant, (seq2->size) * sizeof(Mutant));
    cudaMemcpy(d_seq1, seq1->seq, (seq1->size) * sizeof(char), cudaMemcpyHostToDevice);
    cudaMemcpy(d_seq2, seq2->seq, (seq2->size) * sizeof(char), cudaMemcpyHostToDevice);
    cudaMemcpy(d_weights, weights, 4 * sizeof(double), cudaMemcpyHostToDevice);
    // Launch Kernel
    mutantCreation<<<numOfBlocks, NumOfThreads>>>(d_seq1, d_seq2, size, gpuSize, d_weights, typeMutant, tempMutant);
    // Get the result from the GPU.
    cudaMemcpy(gpuMutant, tempMutant, (tempMutant->size) * sizeof(Mutant), cudaMemcpyDeviceToHost);
    cudaFree(tempMutant->seq);
    cudaFree(d_seq1);
    cudaFree(d_seq2);
    cudaFree(d_weights);
    return gpuMutant;
}
// 1 thread
__global__ void *mutantCreation(Sequence *seq1, Sequence *d_seq2, int gpuSize, double *weights, int typeMutant, Mutant *tempMutant)
{
    int numOfThreadsPerBlock = seq2->size;
    int myIndex = threadIdx.x + blockIdx.x * numOfThreadsPerBlock;

    __shared__ Mutant *tempMutant;
    cudaMalloc((void **)&tempMutant, (seq2->size) * sizeof(Mutant));

    // To compare the first mutant with defult mutant by type (max/min)
    if (typeMutant == 1) //max
        tempMutant->mutantScore = -1111111;
    else //min
        tempMutant->mutantScore = 1111111;

    Mutant *bestMutantPerThread = createMutant(seq1, seq2,  threadIdx.x,  blockIdx.x, typeMutant, weights);

    if (tempMutant->mutantScore == -1111111 || tempMutant->mutantScore == 1111111)
    {
        cudaMalloc((void **)&(tempMutant->seq), (seq2->size) * sizeof(char)); //tempMutant->seq = (char *)malloc(sizeof(char) * seq2->size);
        if (tempMutant->seq == NULL)
        {
            printf("Cannot allocate memory for myMutant");
        }
        cudaMemcpy(tempMutant->seq, bestMutantPerThread->seq, (bestMutantPerThread->size) * sizeof(Mutant), cudaMemcpyDeviceToHost);
        tempMutant->size = bestMutantPerThread->size;
        tempMutant->mutantScore = bestMutantPerThread->mutantScore;
    }
    tempMutant->offset = blockIdx.x + threadIdx.x;
    byMutantType(tempMutant, bestMutantPerThread, typeMutant);
    cudaFree(bestMutantPerThread->seq);
    cudaFree(bestMutantPerThread->seq);

    __syncthreads(); // wait for all threads to finish
}

__device__ Mutant *createMutant(Sequence *seq1, Sequence *seq2, int indexSeq2, int offset, int typeMutant, double *weights)
{
    const char *coservativeGroup[COSER_SIZE] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
    const char *semiCoservativeGroup[SEMI_SIZE] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK",
                                                   "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
    const char letters[LETTERS_SIZE] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K',
                                        'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V',
                                        'W', 'X', 'Y', 'Z', '-'};
    Mutant *myMutant;
    cudaMalloc((void **)&myMutant, sizeof(Mutant)); //Mutant *myMutant = (Mutant *)malloc(sizeof(Mutant));
    if (myMutant == NULL)
    {
        printf("Cannot allocate memory for myMutant");
        return NULL;
    }
    myMutant->seq = NULL;

    for (int i = 0; i < LETTERS_SIZE; i++)
    {
        char letter = letters[i];

        if (checkPairs(&(seq2->seq[indexSeq2]), letter, coservativeGroup, COSER_SIZE))
        {
            if (myMutant->seq == NULL) // Case of compare the first mutant with defult mutant (seq2)
            {
                cudaMalloc((void **)&myMutant->seq, (seq2->size) * sizeof(Mutant)); //myMutant->seq = (char *)malloc(sizeof(char) * seq2->size);

                if (myMutant->seq == NULL)
                {
                    printf("Cannot allocate memory for myMutant");
                    return NULL;
                }
                cudaMemcpy(myMutant->seq, seq2->seq, (seq2->size) * sizeof(char), cudaMemcpyDeviceToHost);
                myMutant->size = seq2->size;
                myMutant->offset = offset;

                if (typeMutant == 1) //max
                    myMutant->mutantScore = -1111111;
                else //min
                    myMutant->mutantScore = 1111111;
            }

            // Second mutant to comapre
            cudaMalloc((void **)&tempMutant, sizeof(Mutant)); //Mutant *tempMutant = (Mutant *)malloc(sizeof(Mutant));

            if (tempMutant == NULL)
            {
                printf("Cannot allocate memory for tempMutant");
                return NULL;
            }

            cudaMalloc((void **)&tempMutant->seq, (myMutant->size) * sizeof(char)); //tempMutant->seq = (char *)malloc(sizeof(char) * myMutant->size);
            if (myMutant->seq == NULL)
            {
                printf("Cannot allocate memory for tempMutant");
                return NULL;
            }
            cudaMemcpy(tempMutant->seq, myMutant->seq, (myMutant->size) * sizeof(char), cudaMemcpyDeviceToHost);
            tempMutant->size = myMutant->size;
            tempMutant->offset = myMutant->offset;
            tempMutant->seq[indexSeq2] = letter;
            tempMutant->mutantScore = calcMutantScore(seq1, tempMutant, weights, coservativeGroup, COSER_SIZE, semiCoservativeGroup, SEMI_SIZE);
            byMutantType(myMutant, tempMutant, typeMutant);
            cudaFree(tempMutant->seq);
            cudaFree(tempMutant);
        }
    }
    return myMutant;
}

__device__ int checkPairs(char *charSeq2, char letter, const char **group, int groupSize)
{
    if (*charSeq2 == letter)
        return 0;
    if (checkInGroupForChange(charSeq2, letter, group, groupSize))
        return 1;
    return 0;
}

__device__ int checkInGroupForChange(char *charSeq2, char letter, const char **group, int groupSize)
{
    for (int i = 0; i < groupSize; i++)
    {
        if (_strchr(group[i], *charSeq2) && _strchr(group[i], letter))
            return 0;
    }
    return 1;
}

__device__ double calcMutantScore(Sequence *seq1, Mutant *myMutant, double *weights, const char **coservativeGroup, int coserSize, const char **semiCoservativeGroup, int semiSize)
{
    int countSignsArr[] = {0, 0, 0, 0}; // {* , : , . , _}

    for (int i = 0; i < myMutant->size; i++)
    {
        char sign = defineSignsMutate(&(seq1->seq[i]), &(myMutant->seq[i]), myMutant->offset, coservativeGroup, coserSize, semiCoservativeGroup, semiSize);
        if (sign == STAR)
            countSignsArr[0] += 1;
        else if (sign == COLON)
            countSignsArr[1] += 1;
        else if (sign == POINT)
            countSignsArr[2] += 1;
        else //sign == SPACE
            countSignsArr[3] += 1;
    }
    return weights[0] * (double)(countSignsArr[0]) - weights[1] * (double)(countSignsArr[1]) - weights[2] * (double)(countSignsArr[2]) - weights[3] * (double)(countSignsArr[3]);
}

__device__ char defineSignsMutate(char *charSeq1, char *mutantChar, int offset, const char **coservativeGroup, int coserSize, const char **semiCoservativeGroup, int semiSize)
{
    if (*(charSeq1 + offset) == *mutantChar)
        return STAR;
    else if (*(charSeq1 + offset) == '-' && *mutantChar != '-')
        return SPACE;
    else if (checkInGroup(charSeq1 + offset, mutantChar, coservativeGroup, coserSize))
        return COLON;
    else if (checkInGroup(charSeq1 + offset, mutantChar, semiCoservativeGroup, semiSize))
        return POINT;
    else
        return SPACE;
}

__device__ int checkInGroup(char *charSeq1, char *mutantChar, const char **group, int groupSize)
{
    for (int i = 0; i < groupSize; i++)
    {
        if (_strchr(group[i], *charSeq1) != NULL && _strchr(group[i], *mutantChar))
            return 1;
    }
    return 0;
}

__device__ void byMutantType(Mutant *mutant1, Mutant *mutant2, int typeMutant)
{
    if (typeMutant) //max
    {
        if (mutant2->mutantScore > mutant1->mutantScore)
        {
            cudaMemcpy(mutant1->seq, mutant2->seq, (mutant2->size) * sizeof(char), cudaMemcpyDeviceToHost);
            mutant1->mutantScore = mutant2->mutantScore;
            mutant1->offset = mutant2->offset;
        }
    }
    else // min
    {
        if (mutant2->mutantScore < mutant1->mutantScore)
        {
            cudaMemcpy(mutant1->seq, mutant2->seq, (mutant2->size) * sizeof(char), cudaMemcpyDeviceToHost);
            mutant1->mutantScore = mutant2->mutantScore;
            mutant1->offset = mutant2->offset;
        }
    }
}

// Checks if char c is in string s on GPU device
__device__ char *_strchr(const char *s, int c) {
	while (*s != (char) c)
		if (!*s++)
			return 0;
	return (char *) s;
}

*/
